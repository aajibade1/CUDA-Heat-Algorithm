#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <unistd.h>
#include <algorithm>
#include <fstream>
#include <cmath>

using namespace std;

//FDM Calculate function -GPU version
__global__ void transferHeat (double *gen, double *hold, int plateSize)
{

    int threadIDX = (blockIdx.x * blockDim.x) + threadIdx.x; // thread id x
    int threadIDY = (blockIdx.y * blockDim.y) + threadIdx.y; // thread id y

    if (threadIDX > 0 && threadIDX < plateSize - 1 && threadIDY > 0 && threadIDY < plateSize - 1)
    {
        gen[threadIDX + (plateSize * threadIDY)] = 0.25 * (hold[(threadIDX - 1) + (plateSize * threadIDY)]
                                                           + hold[(threadIDX + 1) + (plateSize * threadIDY)]
                                                           + hold[threadIDX + (plateSize * (threadIDY + 1))] +
                                                           hold[threadIDX + (plateSize * (threadIDY - 1))]);
    }
}

//copy generated array to holding array function - GPU version
__global__ void copyOver(double *gen, double *hold, int plateSize)
{
    int threadIDX = (blockIdx.x * blockDim.x) + threadIdx.x;
    int threadIDY = (blockIdx.y * blockDim.y)+ threadIdx.y;

    hold[threadIDX + (plateSize * threadIDY)] = gen[threadIDX + (plateSize * threadIDY)];
}

//verify input function
bool is_numeric(const string& strIn, unsigned int& nInputNumber)
{
    //this function checks if the passed input number is a valid ulong value
    //this includes checking for negatives, strings and floating point numbers
    bool bRC = all_of(strIn.begin(), strIn.end(),[](unsigned char c)
                      {
                          return ::isdigit(c);
                      }
    );
    if (bRC)
    {
        nInputNumber = stoi(strIn);//converts string to unsigned long number
        return true;
    }
    else
    {
        return false;
    }
}

int main(int argc, char **argv)
{

    int opt; //for input parameters
    unsigned int N; //N splits for inside data i think
    unsigned int I; //iterations coming soon from a terminal near you
    opterr = 0;

    if (argc > 1)
    {
        if (argv[optind] == NULL || argv[optind + 2] == NULL)
        {
            cout << "Invalid parameter, please check your values." << endl;
            return 0;
        }

        while ((opt = getopt(argc, argv, "N:I:")) != -1)
        {
            bool bIsValid;
            switch (opt)
            {
                case 'N':
                    bIsValid = is_numeric(optarg, N);
                    if (!bIsValid)
                    {
                        cout << "Invalid parameter, please check your values." << endl;
                        return 0;
                    }
                    break;
                case 'I':
                    bIsValid = is_numeric(optarg, I);
                    if (!bIsValid)
                    {
                        cout << "Invalid parameter, please check your values." << endl;
                        return 0;
                    }
                    break;
                case '?':
                    cout << "Invalid parameter, please check your values." << endl;
                    return 0;
                    break;
            }
        }

    }
    else
    {
        cout << "Invalid parameter, please check your values." << endl;
        return 0;
    }

    //cout << "Input N: " << N << endl;
    //cout << "Input I: " << I << endl;
    int plateSize = N + 2; //grid width/height
    int size = sizeof(double) * plateSize * plateSize; //size of an array N*N enough to hold double types
    double *gen = (double *) malloc(size); //host array of gen allocate
    double *hold = (double *) malloc(size); //host array of hold allocate

    //initialize host arrays
    for (int i = 0; i < plateSize; i++)
    {
        for (int j = 0; j < plateSize; j++)
        {
            if ((j > 0.3 * (N + 2 - 1) && j < 0.7 * (N + 2 - 1)) && i == 0) {
                gen[i * plateSize + j] = 100.0;
                hold[i * plateSize + j]  = 100.0;
            }
            else
            {
                gen[i * plateSize + j] = 20.0;
                hold[i * plateSize + j]  = 20.0;
            }
        }
    }

    //allocate device memory
    double *d_gen;
    double *d_hold;

    hipMalloc((void**)&d_gen, size);
    hipMalloc((void**)&d_hold, size);

    //start timing event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Transfer data from host to device memory
    hipMemcpy(d_gen, gen, size, hipMemcpyHostToDevice);
    hipMemcpy(d_hold, hold, size, hipMemcpyHostToDevice);

    //get device info
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int threadsPerBlock=pow(prop.maxThreadsPerBlock, 0.5); //threads in each direction
    dim3 dimBlock(threadsPerBlock, threadsPerBlock );

    // calculate number of blocks along X and Y
    dim3 dimGrid( ceil(double(plateSize)/double(dimBlock.x)), ceil(double(plateSize)/double (dimBlock.y)));

    //cout << "block size: " << ThreadsPerBlock <<  endl;

    hipEventRecord(start); //start the time

    //run kernel functions iteratively
    for (auto iteration = 0; iteration < I; iteration++)
    {
        transferHeat<<<dimGrid, dimBlock>>>(d_gen, d_hold, plateSize);
        copyOver<<<dimGrid, dimBlock>>>(d_gen, d_hold, plateSize);
    }
    hipEventRecord(stop); // stop time
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << setprecision(2) << fixed << milliseconds << endl;

    // Transfer data back to host memory
    hipMemcpy(hold, d_hold, size, hipMemcpyDeviceToHost);
    //cudaThreadSynchronize();


    // Deallocate device memory
    hipFree(d_gen);
    hipFree(d_hold);

    // Deallocate host memory
    delete(gen);
    delete(hold);
    return 0;
}
